#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <omp.h>
#include <iostream>
#include <vector>
#include <algorithm>
#include <thread>
#include <raft/core/nvtx.hpp>
#include <raft/core/device_span.hpp>
#include <raft/core/handle.hpp>
#include <hipcub/hipcub.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/mr/device/cuda_async_memory_resource.hpp>
#include <rmm/mr/device/owning_wrapper.hpp>
#include <rmm/mr/device/pool_memory_resource.hpp>
#include <thrust/host_vector.h>
#include <thrust/mr/allocator.h>
#include <thrust/system/cuda/memory_resource.h>


#define CUDA_CHECK_ERROR(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error in " << __FILE__ << " at line " << __LINE__ << ": " \
                  << hipGetErrorString(err) << std::endl; \
        std::exit(EXIT_FAILURE); \
    } \
} while (0)

inline auto make_async() { return std::make_shared<rmm::mr::cuda_async_memory_resource>(); }
inline auto make_pool()
{
  size_t free_mem, total_mem;
  CUDA_CHECK_ERROR(hipMemGetInfo(&free_mem, &total_mem));
  size_t rmm_alloc_gran = 256;
  double alloc_ratio    = 0.4;
  // allocate 40%
  size_t initial_pool_size = (size_t(free_mem * alloc_ratio) / rmm_alloc_gran) * rmm_alloc_gran;
  return rmm::mr::make_owning_wrapper<rmm::mr::pool_memory_resource>(make_async(),
                                                                     initial_pool_size);
}

template <int TILE_WIDTH, int HISTO_SIZE>
__global__ void computeMedian(raft::device_span<const int> d_matrix, raft::device_span<int> d_median, int width, int height) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < width && y < height) {
        const int index = x + y * width;

        // Declare storage for CUB BlockRadixSort
        typedef hipcub::BlockRadixSort<int,
                                    TILE_WIDTH,
                                    1,
                                    hipcub::NullType,
                                    4,
                                    true,
                                    hipcub::BLOCK_SCAN_WARP_SCANS,
                                    hipSharedMemBankSizeFourByte,
                                    TILE_WIDTH> BlockRadixSort;

        __shared__ typename BlockRadixSort::TempStorage temp_storage;

        int thread_keys[1];
        thread_keys[0] = d_matrix[index];

        // Perform block-level radix sort
        BlockRadixSort(temp_storage).Sort(thread_keys);

        if (threadIdx.x == TILE_WIDTH / 2 && threadIdx.x == TILE_WIDTH / 2) {
            d_median[blockIdx.x + blockIdx.y * gridDim.x] = thread_keys[0];
        }
    }
}

int main() {
    constexpr auto TILE_WIDTH = 32;
    constexpr auto HISTO_SIZE = 256;
    constexpr auto NB_TILE_X = 250;
    constexpr auto NB_TILE_Y = NB_TILE_X;
    constexpr auto MATRIX_LEGNTH = TILE_WIDTH * NB_TILE_X;
    constexpr auto MATRIX_SIZE = MATRIX_LEGNTH * MATRIX_LEGNTH;
    constexpr auto NB_IMAGES = 3;
    constexpr auto INIT_VALUE = 4;

    auto memory_resource = make_pool();
    rmm::mr::set_current_device_resource(memory_resource.get());

    using host_pinned_vector = thrust::host_vector<int,
                      thrust::mr::stateless_resource_allocator<
                        int,
                        thrust::system::cuda::universal_host_pinned_memory_resource>>;

    std::vector<host_pinned_vector> h_matrices(NB_IMAGES, host_pinned_vector(MATRIX_SIZE, 4));
    std::vector<host_pinned_vector> h_medians(NB_IMAGES, host_pinned_vector(NB_TILE_X * NB_TILE_Y));

    std::vector<std::thread> threads;

    raft::common::nvtx::push_range("Images compute");

#pragma omp parallel for
    for (int i = 0; i < NB_IMAGES; ++i)
    {
        raft::common::nvtx::range fun_scope("Image compute");

        const raft::handle_t handle{};

        int thread_id = omp_get_thread_num();

        const host_pinned_vector& d_matrix = h_matrices[thread_id];
        host_pinned_vector& d_median = h_medians[thread_id];

        raft::common::nvtx::push_range("Kernel");

        // Launch kernel
        dim3 blockSize(TILE_WIDTH, TILE_WIDTH);
        dim3 gridSize((MATRIX_LEGNTH + blockSize.x - 1) / blockSize.x, (MATRIX_LEGNTH + blockSize.y - 1) / blockSize.y);
        computeMedian<TILE_WIDTH, HISTO_SIZE><<<gridSize, blockSize, 0, handle.get_stream()>>>(raft::device_span<const int>{thrust::raw_pointer_cast(d_matrix.data()), d_matrix.size()}, raft::device_span<int>{thrust::raw_pointer_cast(d_median.data()), d_median.size()}, MATRIX_LEGNTH, MATRIX_LEGNTH);
        CUDA_CHECK_ERROR(hipGetLastError());

        raft::common::nvtx::pop_range();


        raft::common::nvtx::pop_range();

        CUDA_CHECK_ERROR(hipStreamSynchronize(handle.get_stream()));
    }

    CUDA_CHECK_ERROR(hipDeviceSynchronize());

    raft::common::nvtx::pop_range();

    for (int image = 0; image < NB_IMAGES; ++image)
    {
        if (!std::all_of(h_medians[image].cbegin(), h_medians[image].cend(), [INIT_VALUE](int i){ return i == INIT_VALUE; }))
        {
            std::cout << "Value should be " << INIT_VALUE << std::endl;
            for (int i = 0; i <= 6; ++i)
                std::cout << h_medians[image][i] << " ";
            std::cout << std::endl;
            return -1;
        }
    }

    std::cout << "All good" << std::endl;

    return 0;
}